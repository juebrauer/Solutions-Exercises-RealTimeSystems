
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <stdio.h>


// Note: for profiling use
//  nvcc add_gpu.cu -o add_gpu
//  sudo nvprof --unified-memory-profiling off ./add_gpu
//
// Hint: 
// https://stackoverflow.com/questions/36970646/nvprof-not-picking-up-any-api-calls-or-kernels

const int VARIANT_NR = 2;

// Kernel function to add the elements of two arrays

// Variant 1
// SLOW!
__global__
void add1(int n, float *x, float *y)
{
  for (int i = 0; i < n; i++)
    y[i] = x[i] + y[i];
}


// Variant 2
// FAST!
__global__
void add2(int n, float *x, float *y)
{
  int index = threadIdx.x;
  int stride = blockDim.x;
  //std::cout << "Hello from thread " << threadIdx.x <<
  //            " in block " << blockDim.x << "\n";
  //printf("threadIdx.x=%d, threadIdx.y=%d, blockDim.x=%d, blockDim.y=%d\n",
  // threadIdx.x, threadIdx.y, blockDim.x, blockDim.y);
  for (int i = index; i < n; i += stride)
      y[i] = x[i] + y[i];
}


// Variant 3
// FAST!
__global__
void add3(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  /*
  printf("threadIdx.x=%d, threadIdx.y=%d, blockIdx.x=%d, blockIdy.y=%d " \
         "blockDim.x=%d, blockDim.y=%d, gridDim.x=%d\n",
         threadIdx.x, threadIdx.y,
         blockIdx.x, blockIdx.y,
         blockDim.x, blockDim.y, gridDim.x);
  */
  
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];

}


int main(void)
{
  int N = 1<<20;
  float *x, *y;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Choose selected kernel to add
  // two vectors of 1M elements each
  // on GPU
  int NrBlocks, NrThreads;

  if (VARIANT_NR==1)
  {
    printf("Launching kernel add1\n");
    NrBlocks = 1;
    NrThreads = 1;
    printf("NrBlocks=%d, NrThreads=%d\n", NrBlocks, NrThreads);
    add1<<<NrBlocks, NrThreads>>>(N, x, y);
  }

  if (VARIANT_NR==2)
  {
    printf("Launching kernel add2\n");
    NrBlocks = 1;
    NrThreads = 256;
    printf("NrBlocks=%d, NrThreads=%d\n", NrBlocks, NrThreads);
    add2<<<NrBlocks, NrThreads>>>(N, x, y);
  }

  if (VARIANT_NR==3)
  {
    printf("Launching kernel add3\n");
    NrThreads = 32;
    NrBlocks = N/NrThreads;
    if (N%NrThreads!=0)
      NrBlocks++;
    printf("NrBlocks=%d, NrThreads=%d\n", NrBlocks, NrThreads);
    add3<<<NrBlocks, NrThreads>>>(N, x, y);
  }


  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  for (int i=0; i<10; i++)
    printf("i=%d: %f\n", i, y[i]);
  for (int i=N-10; i<N; i++)
    printf("i=%d: %f\n", i, y[i]);

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);

  return 0;
}
